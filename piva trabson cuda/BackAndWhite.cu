#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "BlackAndWhite.h"
#include "Grayscale.h"

__global__ void black_and_white_CUDA(unsigned char* imagem, int canais);

void black_and_white(unsigned char* imagem, int altura, int largura, int canais) {
	gray_scale(imagem, altura, largura, canais);
	unsigned char* dev_imagem = NULL;

	hipMalloc((void**)&dev_imagem, altura * largura * canais);

	hipMemcpy(dev_imagem, imagem, altura * largura * canais, hipMemcpyHostToDevice);

	dim3 Grid_Image(largura, altura);
	black_and_white_CUDA <<< Grid_Image, 1 >>> (dev_imagem, canais);

	hipMemcpy(imagem, dev_imagem, altura * largura * canais, hipMemcpyDeviceToHost);
	hipFree(dev_imagem);
}

__global__ void black_and_white_CUDA(unsigned char* imagem, int canais) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = (x + y * gridDim.x) * canais;

	imagem[idx + 0] = imagem[idx + 0] > 127 ? 255 : 0;
	imagem[idx + 1] = imagem[idx + 1] > 127 ? 255 : 0;
	imagem[idx + 2] = imagem[idx + 2] > 127 ? 255 : 0;
}