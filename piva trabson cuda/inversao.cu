#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Inversao.h"

__global__ void inversao_CUDA(unsigned char* imagem, int canais);

void inversao(unsigned char* imagem, int altura, int largura, int canais) {
	unsigned char* dev_imagem = NULL;

	hipMalloc((void**)&dev_imagem, altura * largura * canais);

	hipMemcpy(dev_imagem, imagem, altura * largura * canais, hipMemcpyHostToDevice);

	dim3 Grid_Image(largura, altura);
	inversao_CUDA <<<Grid_Image, 1>>> (dev_imagem, canais);

	hipMemcpy(imagem, dev_imagem, altura * largura * canais, hipMemcpyDeviceToHost);
	hipFree(dev_imagem);
}

__global__ void inversao_CUDA(unsigned char* imagem, int canais) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = (x + y * gridDim.x) * canais;

	for (int i = 0; i < canais; i++) {
		imagem[idx + i] = 255 - imagem[idx + i];
	}
}