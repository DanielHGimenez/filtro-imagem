#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Grayscale.h"

__global__ void gray_scale_CUDA(unsigned char* imagem, int canais);

void gray_scale(unsigned char* imagem, int altura, int largura, int canais) {
	unsigned char* dev_imagem = NULL;

	hipMalloc((void**)&dev_imagem, altura * largura * canais);

	hipMemcpy(dev_imagem, imagem, altura * largura * canais, hipMemcpyHostToDevice);

	dim3 Grid_Image(largura, altura);
	gray_scale_CUDA <<< Grid_Image, 1 >>> (dev_imagem, canais);

	hipMemcpy(imagem, dev_imagem, altura * largura * canais, hipMemcpyDeviceToHost);
	hipFree(dev_imagem);
}

__global__ void gray_scale_CUDA(unsigned char* imagem, int canais) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = (x + y * gridDim.x) * canais;

	char gray_color = (imagem[idx + 0] + imagem[idx + 1] + imagem[idx + 2]) / 3;

	imagem[idx + 0] = gray_color;
	imagem[idx + 1] = gray_color;
	imagem[idx + 2] = gray_color;
}