#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Espelhamento.h"

__global__ void espelhamento_CUDA(unsigned char* imagem, int canais);

void espelhamento(unsigned char* imagem, int altura, int largura, int canais) {
	unsigned char* dev_imagem = NULL;

	hipMalloc((void**)&dev_imagem, altura * largura * canais);

	hipMemcpy(dev_imagem, imagem, altura * largura * canais, hipMemcpyHostToDevice);

	dim3 Grid_Image(largura / 2, altura);
	espelhamento_CUDA <<< Grid_Image, 1 >>> (dev_imagem, canais);

	hipMemcpy(imagem, dev_imagem, altura * largura * canais, hipMemcpyDeviceToHost);
	hipFree(dev_imagem);
}

__global__ void espelhamento_CUDA(unsigned char* imagem, int canais) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = (x + y * (gridDim.x * 2)) * canais;
	int idx2 = ((gridDim.x * 2) - x + y * (gridDim.x * 2)) * canais;

	for (int i = 0; i < canais; i++) {
		char aux = imagem[idx + i];
		imagem[idx + i] = imagem[idx2 + i];
		imagem[idx2 + i] = aux;
	}
}