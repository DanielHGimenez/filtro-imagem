#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Brightness.h"

__global__ void brightness_CUDA(unsigned char* imagem, int brightness, int canais);

void brightness(unsigned char* imagem, int brightness, int altura, int largura, int canais) {
	unsigned char* dev_imagem = NULL;

	hipMalloc((void**)&dev_imagem, altura * largura * canais);

	hipMemcpy(dev_imagem, imagem, altura * largura * canais, hipMemcpyHostToDevice);

	dim3 Grid_Image(largura, altura);
	brightness_CUDA <<< Grid_Image, 1 >>> (dev_imagem, brightness, canais);

	hipMemcpy(imagem, dev_imagem, altura * largura * canais, hipMemcpyDeviceToHost);
	hipFree(dev_imagem);
}

__global__ void brightness_CUDA(unsigned char* imagem, int brightness, int canais) {
	int x = blockIdx.x;
	int y = blockIdx.y;

	int idx = (x + y * gridDim.x) * canais;

	for (int i = 0; i < canais; i++) {
		int pixel = imagem[idx + i] + brightness;
		
		if (pixel > 255)
			pixel = 255;
		else if (pixel < 0)
			pixel = 0;

		imagem[idx + i] = pixel;
	}
}